#include "hip/hip_runtime.h"

#include <time.h>
#include "cudaKernelTask.h"

#ifndef checkCudaErrorsKernel
#define checkCudaErrorsKernel(err) __checkCudaErrors(err, __FILE__, __LINE__)
// TODO: Do we need replace all the drv api to cudaruntime api?

inline void __checkCudaErrors(hipError_t err, const char* file, const int line) {
  if (hipSuccess != err) {
    const char* errorStr = NULL;
    hipDrvGetErrorString(err, &errorStr);
    fprintf(
        stderr,
        "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, "
        "line %i.\n",
        err,
        errorStr,
        file,
        line);
    exit(EXIT_FAILURE);
  }
}
#endif

__global__ void matrixMultiplyKernel(double* dataC, double* dataA, double* dataB, unsigned int matrixDim) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  double value = 0.0f;
  if (i > matrixDim) {
    i = i % matrixDim;
  }

  if (j > matrixDim) {
    j = j % matrixDim;
  }

  for (unsigned int k = 0; k < matrixDim; ++k) {
    value += dataA[k + i * matrixDim] * dataB[j + k * matrixDim];
  }
  dataC[j + i * matrixDim] = value;
}

void fillMatrixWithRandomValues(double* matrix, unsigned int matrixDim) {
  time_t t;
  unsigned int i, j;
  srand((unsigned int)time(&t));

  for (i = 0; i < matrixDim; ++i) {
    for (j = 0; j < matrixDim; ++j) {
      matrix[j + i * matrixDim] = (float)(rand() & 0xFF) / 10.0f;
      ;
    }
  }
}

void initMatrix(
    double** hptrA,
    double** hptrB,
    double** hptrC,
    hipDeviceptr_t* dptrA,
    hipDeviceptr_t* dptrB,
    hipDeviceptr_t* dptrC,
    int matrixDim) {
  unsigned int matrixSize = matrixDim * matrixDim;
  size_t byteSize = matrixSize * sizeof(double);

  checkCudaErrorsKernel(hipMalloc(dptrA, byteSize));
  checkCudaErrorsKernel(hipMalloc(dptrB, byteSize));
  checkCudaErrorsKernel(hipMalloc(dptrC, byteSize));
  checkCudaErrorsKernel(hipMemsetD8(*dptrC, 0, byteSize));

  *hptrA = new double[matrixSize];
  *hptrB = new double[matrixSize];
  *hptrC = new double[matrixSize];

  fillMatrixWithRandomValues(*hptrA, matrixDim);
  fillMatrixWithRandomValues(*hptrB, matrixDim);

  checkCudaErrorsKernel(hipMemcpyHtoD(*dptrA, *hptrA, byteSize));
  checkCudaErrorsKernel(hipMemcpyHtoD(*dptrB, *hptrB, byteSize));
}

void runMultiplyMatrix(
    hipDeviceptr_t dptrA, hipDeviceptr_t dptrB, hipDeviceptr_t dptrC, hipStream_t stream, int matrixDim, int loopCount) {
  dim3 threads(32, 32);
  dim3 grid(matrixDim / threads.x, matrixDim / threads.y);

  for (int i = 0; i < loopCount; i++) {
    matrixMultiplyKernel<<<grid, threads, 0, stream>>>(
        reinterpret_cast<double*>(dptrC),
        reinterpret_cast<double*>(dptrA),
        reinterpret_cast<double*>(dptrB),
        matrixDim);
  }
}

void FreeMatrix(double* hptrA, double* hptrB, double* hptrC, hipDeviceptr_t dptrA, hipDeviceptr_t dptrB, hipDeviceptr_t dptrC) {
  checkCudaErrorsKernel(hipFree(dptrA));
  checkCudaErrorsKernel(hipFree(dptrB));
  checkCudaErrorsKernel(hipFree(dptrC));
  free(hptrA);
  free(hptrB);
  free(hptrC);
  return;
}
